#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void kernel( int *a, int dimx, int dimy ) {
	int ix = blockIdx.x*blockDim.x + threadIdx.x; 
	int iy = blockIdx.y*blockDim.y + threadIdx.y; 
	int idx = iy*dimx + ix;
	a[idx] = a[idx]+1;
}

int main() {
	int dimx = 16, dimy = 16;
	int num_bytes = dimx*dimy*sizeof(int);
	int *d_a=0, *h_a=0; // device and host pointers h_a = (int*)malloc(num_bytes);
	
	hipMalloc( (void**)&d_a, num_bytes );
	
	if( 0==h_a || 0==d_a ) {
		printf("couldn't allocate memory\n"); return 1;
	}
	hipMemset( d_a, 0, num_bytes );
	
	dim3 grid, block;
	block.x = 4; block.y = 4;
	grid.x = dimx / block.x;
	grid.y = dimy / block.y;
	kernel<<<grid, block>>>( d_a, dimx, dimy );
	
	hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
	
	free( h_a ); hipFree( d_a ); 
	
	return 0;
}