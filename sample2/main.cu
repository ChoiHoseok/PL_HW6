#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__
void cudaMultVectorsKernel(int N, float *x, float *y, float *z)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N) {
    z[idx] = x[idx] * y[idx];
  }
  // idx = idx + blockDim.x * gridDim.x; // we will discuss this later...
}

// extern "C" is necessary because nvcc uses c++ compiler to compile cuda code
// hence applies name mangling. Because we use gcc for linking, we should 
// prevent name mangling.
extern "C"
void runKernel(int N, float *x, float *y, float *z) {
    cudaMultVectorsKernel<<<(N+511)/512, 512>>>(N, x, y, z);
}

int main() {
    const int N = 1<<20;
    float *x, *y, *z, *dx, *dy, *dz;
	
	//printf("hello world\n");
    hipMalloc((void**)&dx, N*sizeof(float));
    hipMalloc((void**)&dy, N*sizeof(float));
    hipMalloc((void**)&dz, N*sizeof(float));

    // init array x, y
    for (int i=0; i<N; i++) {
        x[i] = 2.3f*i;
        y[i] = 4.1f*i;
    }

    hipMemcpy(dx, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, N*sizeof(float), hipMemcpyHostToDevice);

    runKernel(N, dx, dy, dz);

    hipMemcpy(z, dz, N*sizeof(float), hipMemcpyDeviceToHost);
	//printf("hello world\n");  
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
	//printf("hello world\n");
    free(x);
    free(y);
    free(z);
	//printf("hello world\n");
    return 0;
}


