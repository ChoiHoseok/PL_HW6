
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < n)
		c[index] = a[index] + b[index];
}

void random_ints(int* a);
int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	int i;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);


	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	random_ints(a);
	random_ints(b);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	for(i = 0; i < 512; i++){
		printf("%d ",c[i]);
		if(i%5 == 0)
			printf("\n");
	}
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_a);
	hipFree(d_a);

	return 0;
}

void random_ints(int* a)
{
	int i;
	for ( i = 0; i < 512; ++i)
		a[i] = rand();
}


